//==============================================================
// Copyright © 2020 Intel Corporation
//
// SPDX-License-Identifier: MIT
// =============================================================
//
// Bitonic Sort: this algorithm converts a randomized sequence of numbers into
// a bitonic sequence (two ordered sequences), and then merge these two ordered
// sequences into a ordered sequence. Bitonic sort algorithm is briefly
// described as followed:
//
// - First, it decomposes the randomized sequence of size 2**n into 2**(n-1)
// pairs where each pair consists of 2 consecutive elements. Note that each pair
// is a bitonic sequence.
// - Step 0: for each pair (sequence of size 2), the two elements are swapped so
// that the two consecutive pairs form  a bitonic sequence in increasing order,
// the next two pairs form the second bitonic sequence in decreasing order, the
// next two pairs form the third bitonic sequence in  increasing order, etc, ...
// . At the end of this step, we have 2**(n-1) bitonic sequences of size 2, and
// they follow an order increasing, decreasing, increasing, .., decreasing.
// Thus, they form 2**(n-2) bitonic sequences of size 4.
// - Step 1: for each new 2**(n-2) bitonic sequences of size 4, (each new
// sequence consists of 2 consecutive previous sequences), it swaps the elements
// so that at the end of step 1, we have 2**(n-2) bitonic sequences of size 4,
// and they follow an order: increasing, decreasing, increasing, ...,
// decreasing. Thus, they form 2**(n-3) bitonic sequences of size 8.
// - Same logic applies until we reach the last step.
// - Step n: at this last step, we have one bitonic sequence of size 2**n. The
// elements in the sequence are swapped until we have a sequence in increasing
// oder.
//
// In this implementation, a randomized sequence of size 2**n is given (n is a
// positive number). Unified Shared Memory (USM) is used for data management. At
// each stage, a part of step, the host redefines the ordered sequenes and sends
// data to the kernel. The kernel swaps the elements accordingly in parallel.
//
#include <math.h>
#include <chrono>
#include <iostream>
#include <limits>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 256

__global__ void bs (const int seq_len, const int two_power, int *a) {

  int i = blockDim.x * blockIdx.x + threadIdx.x;

  // Assign the bitonic sequence number.
  int seq_num = i / seq_len;

  // Variable used to identified the swapped element.
  int swapped_ele = -1;

  // Because the elements in the first half in the bitonic
  // sequence may swap with elements in the second half,
  // only the first half of elements in each sequence is
  // required (seq_len/2).
  int h_len = seq_len / 2;

  if (i < (seq_len * seq_num) + h_len) swapped_ele = i + h_len;

  // Check whether increasing or decreasing order.
  int odd = seq_num / two_power;

  // Boolean variable used to determine "increasing" or
  // "decreasing" order.
  bool increasing = ((odd % 2) == 0);

  // Swap the elements in the bitonic sequence if needed
  if (swapped_ele != -1) {
    if (((a[i] > a[swapped_ele]) && increasing) ||
	((a[i] < a[swapped_ele]) && !increasing)) {
      int temp = a[i];
      a[i] = a[swapped_ele];
      a[swapped_ele] = temp;
    }
  }
}

void ParallelBitonicSort(int data_gpu[], int n) {

  // n: the exponent used to set the array size. Array size = power(2, n)
  int size = pow(2, n);

  int *a;
  hipMalloc((void**)&a, sizeof(int) * size);
  hipMemcpy(a, data_gpu, sizeof(int) * size, hipMemcpyHostToDevice);
  
  // step from 0, 1, 2, ...., n-1
  for (int step = 0; step < n; step++) {
    // for each step s, stage goes s, s-1, ..., 0
    for (int stage = step; stage >= 0; stage--) {
      // In each state, construct a number (num_seq) of bitonic sequences of
      // size seq_len (2, 4, ...) num_seq stores the number of bitonic sequences
      // at each stage. seq_len stores the length of the bitonic sequence at
      // each stage.
      int seq_len = pow(2, stage + 1);
#if DEBUG
      int num_seq = pow(2, (n - stage - 1));  // Used for debug purpose.
      std::cout << "step num:" << step << " stage num:" << stage
                << " num_seq:" << num_seq << "(" << seq_len << ") => ";
#endif
      // Constant used in the kernel: 2**(step-stage).
      int two_power = 1 << (step - stage);
      bs<<< dim3(size/BLOCK_SIZE), dim3(BLOCK_SIZE) >>> (seq_len, two_power, a);
    }  // end stage
  } // end step
  hipMemcpy(data_gpu, a, sizeof(int) * size, hipMemcpyDeviceToHost);
  hipFree( a );
}

// Loop over the bitonic sequences at each stage in serial.
void SwapElements(int step, int stage, int num_sequence, int seq_len,
                  int *array) {
  for (int seq_num = 0; seq_num < num_sequence; seq_num++) {
    int odd = seq_num / (pow(2, (step - stage)));
    bool increasing = ((odd % 2) == 0);

    int h_len = seq_len / 2;

    // For all elements in a bitonic sequence, swap them if needed
    for (int i = seq_num * seq_len; i < seq_num * seq_len + h_len; i++) {
      int swapped_ele = i + h_len;

      if (((array[i] > array[swapped_ele]) && increasing) ||
          ((array[i] < array[swapped_ele]) && !increasing)) {
        int temp = array[i];
        array[i] = array[swapped_ele];
        array[swapped_ele] = temp;
      }
    }  // end for all elements in a sequence
  }    // end all sequences
}

// Function sorts an array in serial using bitonic sort algorithm. The size of
// the array is indicated by the exponent n: the array size is 2 ** n.
inline void BitonicSort(int a[], int n) {
  // n: the exponent indicating the array size = 2 ** n.

  // step from 0, 1, 2, ...., n-1
  for (int step = 0; step < n; step++) {
    // for each step s, stage goes s, s-1,..., 0
    for (int stage = step; stage >= 0; stage--) {
      // Sequences (same size) are formed at each stage.
      int num_sequence = pow(2, (n - stage - 1));
      // The length of the sequences (2, 4, ...).
      int sequence_len = pow(2, stage + 1);

      SwapElements(step, stage, num_sequence, sequence_len, a);
    }
  }
}

// Function showing the array.
void DisplayArray(int a[], int array_size) {
  for (int i = 0; i < array_size; ++i) std::cout << a[i] << " ";
  std::cout << "\n";
}

void Usage(std::string prog_name, int exponent) {
  std::cout << " Incorrect parameters\n";
  std::cout << " Usage: " << prog_name << " n k \n\n";
  std::cout << " n: Integer exponent presenting the size of the input array. "
               "The number of element in\n";
  std::cout << "    the array must be power of 2 (e.g., 1, 2, 4, ...). Please "
               "enter the corresponding\n";
  std::cout << "    exponent betwwen 0 and " << exponent - 1 << ".\n";
  std::cout << " k: Seed used to generate a random sequence.\n";
}

int main(int argc, char *argv[]) {
  int n, seed, size;
  int exp_max = log2(std::numeric_limits<int>::max());

  // Read parameters.
  try {
    n = std::stoi(argv[1]);

    // Verify the boundary of acceptance.
    if (n < 0 || n >= exp_max) {
      Usage(argv[0], exp_max);
      return -1;
    }

    seed = std::stoi(argv[2]);
    size = pow(2, n);
  } catch (...) {
    Usage(argv[0], exp_max);
    return -1;
  }

  std::cout << "\nArray size: " << size << ", seed: " << seed << "\n";

  // Memory allocated for host access only.
  int *data_cpu = (int *)malloc(size * sizeof(int));

  // Memory allocated to store gpu results
  int *data_gpu = (int *)malloc(size * sizeof(int));

  // Initialize the array randomly using a seed.
  srand(seed);

  for (int i = 0; i < size; i++) {
    data_gpu[i] = data_cpu[i] = rand() % 1000;
  }


#if DEBUG
  std::cout << "\ndata before:\n";
  DisplayArray(data_gpu, size);
#endif

  auto start = std::chrono::steady_clock::now();

  ParallelBitonicSort(data_gpu, n);

  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  std::cout << "Parallel bitonic time " << (time * 1e-9f) << " (s)\n";

#if DEBUG
  std::cout << "\ndata after sorting using parallel bitonic sort:\n";
  DisplayArray(data_gpu, size);
#endif

  // Bitonic sort in CPU (serial)
  BitonicSort(data_cpu, n);

  // Verify both bitonic sort algorithms in kernel and in CPU.
  bool pass = true;
  for (int i = 0; i < size - 1; i++) {
    // Validate the sequence order is increasing in both kernel and CPU.
    if ((data_gpu[i] > data_gpu[i + 1]) || (data_gpu[i] != data_cpu[i])) {
      pass = false;
      break;
    }
  }

  // Clean CPU memory.
  free(data_cpu);
  free(data_gpu);

  if (!pass) {
    std::cout << "\nFailed!\n";
    return -2;
  }

  std::cout << "\nSuccess!\n";
  return 0;
}
